#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#define GLEW_STATIC
#include <GL/glew.h>
// GLFW
#include <GLFW/glfw3.h>
//CUDA
#include "hip/hip_runtime.h"
#include ""
// Other includes
#include "Shader.h"
#include "Particle.h"
#include "Utility.h"

#define GRAVITY 2000 //some constants I need
#define DRAG 10
#define RESTITUTION_COEFFICIENT 1 // how much energy must be absorbed when bouncing off a wall
#define INITIAL_DISTANCE 0.01 // how far particles are one another initially
#define MOUSE_FORCE -20000
#define CHUNK_NB 10 // separating particles in smaller chunks to avoid having HUGE arrays (1 billion particles) : else we might face stack overflow or framerate drops. To understand the code faster, you can think that CHUNK_NB = 1
#define VERTEX_CHUNK 100000 // how much particles are in each chunk
#define PARTICLE_SIZE 10000//how much particles totally

// Function prototypes
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);
void mouse_callback(GLFWwindow* window, double xpos, double ypos);
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Window dimensions
const GLuint WIDTH = 800, HEIGHT = 600;
GLfloat deltaTime = 0.0f;
GLfloat lastFrame = 0.0f;
GLfloat FPS = 0.0f;
const int particleRow = 100;
const int particleCol = PARTICLE_SIZE/ particleRow;
vec2 mousePos = vec2(0, 0);
bool LMB = false; // is left mouse button hit ?
float dt = 0.003;
using namespace std;
using namespace glm;
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	////////////////////////////////////////////////
	//
	//	CUDA part
	//
	/////////////////////////////////////////////////
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	////////////////////////////////////////////////
	//
	//	GL part
	//
	/////////////////////////////////////////////////
	// Init GLFW
	vector<Particle> particles; // an array storing Particle instances (that we'll move)
	int particleSize = PARTICLE_SIZE; // avoid repeating particles.size() during the for loop to save some time (remember that the for loop is done 1 billion time per frame !
	for (int i(0); i < particleRow; i++) // storing Particle instances in the particles array
	{
		for (int j(0); j < particleCol; j++)
		{
			Particle particle; // see Particle.h and Particle.cpp
			particle.setPosition(vec2(j*INITIAL_DISTANCE, i*INITIAL_DISTANCE)); // we place the particles in a square shape
			particle.setMass(10);
			particles.push_back(particle);
		}
	}

	glfwInit();
	// Set all the required options for GLFW
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	// Create a GLFWwindow object that we can use for GLFW's functions
	GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "LearnOpenGL", nullptr, nullptr);
	glfwMakeContextCurrent(window);

	// Set the required callback functions
	glfwSetKeyCallback(window, key_callback);
	glfwSetCursorPosCallback(window, mouse_callback);
	glfwSetMouseButtonCallback(window, mouse_button_callback);
	// Set this to true so GLEW knows to use a modern approach to retrieving function pointers and extensions
	glewExperimental = GL_TRUE;
	// Initialize GLEW to setup the OpenGL Function pointers
	glewInit();

	// Define the viewport dimensions
	glViewport(0, 0, WIDTH, HEIGHT);


	// Build and compile our shader program
	Shader ourShader("vs.txt", "fs.txt");


	// Set up vertex data (and buffer(s)) and attribute pointers
	GLfloat* vertices =  new GLfloat [PARTICLE_SIZE * 6];
	/*GLfloat vertices[] = {
		// Positions         // Colors
		0.5f, -0.5f, 0.0f,   1.0f, 0.0f, 0.0f,  // Bottom Right
		-0.5f, -0.5f, 0.0f,   0.0f, 1.0f, 0.0f,  // Bottom Left
		0.0f,  0.5f, 0.0f,   0.0f, 0.0f, 1.0f   // Top 
	};
	*/
	GLuint VBO, VAO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	// Bind the Vertex Array Object first, then bind and set vertex buffer(s) and attribute pointer(s).
	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, PARTICLE_SIZE * 6 * sizeof(GLfloat), vertices, GL_DYNAMIC_DRAW);
	//cout << "PARTICLE_SIZE * 6 * sizeof(float):" << PARTICLE_SIZE * 6 * sizeof(float) << endl;
	// Position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(GLfloat), (GLvoid*)0);
	glEnableVertexAttribArray(0);
	// Color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(GLfloat), (GLvoid*)(3 * sizeof(GLfloat)));
	glEnableVertexAttribArray(1);

	glBindVertexArray(0); // Unbind VAO

	//glEnable(GL_POINT_SMOOTH); // allow to have rounded dots
	//glEnable(GL_BLEND);
	//glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glPointSize(2);
	// Game loop
	while (!glfwWindowShouldClose(window))
	{
		// Set frame time
		GLfloat currentFrame = glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		FPS = 60.0f / deltaTime;
		if (deltaTime >= 1.0)
		{
			lastFrame = currentFrame;
			cout << FPS << endl;
			cout << particles[0].getSpeed().x << ", " << particles[0].getSpeed().y << ":" << sqrt(pow(particles[0].getSpeed().x, 2) + pow(particles[0].getSpeed().y, 2)) << endl;
		}
		// Check if any events have been activiated (key pressed, mouse moved etc.) and call corresponding response functions
		glfwPollEvents();

		for (int i(0); i < particleSize; i++) // now, each frame, we want to update each particle's position according to the newton's law, color according to its speed, and maybe make it collide with walls (this for loop is executed 1 billion times each frame)
		{
			//particles[i].addForce(Vector2f(0,GRAVITY)) ; // example for adding gravity force
			particles[i].addForce((vec2(mousePos - particles[i].getPosition()) * (float)(LMB * 10000 / pow(Distance(mousePos, particles[i].getPosition())+5, 2)))); 
			// if the user clicks we add a force proportionnal to the inverse of the distance squared
			particles[i].addForce(-particles[i].getSpeed()*(float)DRAG); 
			// we add a drag force proportionnal to the speed

		    //previousPosition = particles[i].getPosition() ; // uncomment this line if you want to perform collision detection
			particles[i].updatePosition(dt); // we update the position of the particle according to the Newton's law (see Particle.h and Particle.cpp)

			particles[i].clearForce(); // we don't want forces to add over time so we clear them before adding them the next frame

									   /*for(int j(0) ; j < wallPoints.size() ; j+=2) // uncomment these lines if you want to perform collision detection
									   {
									   if(determinant(wallPoints[j+1] - wallPoints[j], wallPoints[j+1]-particles[i].getPosition())*determinant(wallPoints[j+1] - wallPoints[j], wallPoints[j+1]-previousPosition)<0) // if we crossed a wall during this frame
									   {
									   Vector2f v = wallPoints[j+1] - wallPoints[j] ; // vector directing the wall
									   Vector2f n = Vector2f(-v.y,v.x) ; // vector normal to the wall
									   n/=Norm(n) ; // we want the normal vector to be a unit vector (length = 1)
									   particles[i].setPosition(previousPosition) ; // we put the particle in its previous position (in front of the wall, since it passed it)
									   float j = -(1+RESTITUTION_COEFFICIENT)*dotProduct(particles[i].getSpeed(), n) ; // we compute the speed after bouncing off

									   particles[i].setSpeed(particles[i].getSpeed() + j*n) ; // we change the speed
									   }
									   }*/

		}
		for (int i(0); i < particleSize; i++) // we convert Vector2f positions to the OpenGL's way of storing positions : static arrays of floats
		{
			
			vertices[ i*6 ] = particles[i].getPosition().x;
			vertices[i*6 + 1] = particles[i].getPosition().y;
			vertices[i*6 + 2] = 0.0f;
			vertices[i*6 + 3] = clamp(100 * Norm(particles[i].getSpeed()), 0, 255);
			vertices[i*6 + 4] = clamp(255-100*Norm(particles[i].getSpeed()), 0, 255); // we change the particle's colors according to their speed
			vertices[i*6 + 5] = 0.0f;
			
		}
		glBindVertexArray(VAO);
		glBindBuffer(GL_ARRAY_BUFFER, VBO);
		glBufferData(GL_ARRAY_BUFFER, PARTICLE_SIZE * 6 * sizeof(GLfloat), vertices, GL_DYNAMIC_DRAW);

		// Render
		// Clear the colorbuffer
		glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		// Draw the triangle
		ourShader.Use();
		
		glDrawArrays(GL_POINTS, 0, particleSize);
		//glDrawArrays(GL_TRIANGLES, 0, 3);
		
		glBindVertexArray(0);

		// Swap the screen buffers
		glfwSwapBuffers(window);
	}
	// Properly de-allocate all resources once they've outlived their purpose
	delete[] vertices;
	glDeleteVertexArrays(1, &VAO);
	glDeleteBuffers(1, &VBO);
	// Terminate GLFW, clearing any resources allocated by GLFW.
	glfwTerminate();

	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

// Is called whenever a key is pressed/released via GLFW
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
{
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
		glfwSetWindowShouldClose(window, GL_TRUE);
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{
	mousePos = vec2(2*xpos/WIDTH - 1 ,-2*ypos/HEIGHT + 1 );
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS)
	{
		LMB = true;
		cout << "mousePos:" << mousePos.x << ", " << mousePos.y << endl;

	}
	else
	{
		LMB = false;
	}
}